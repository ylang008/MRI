#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#define PI   3.1415926535897932384626433832795029f
#define PIx2 6.2831853071795864769252867665590058f

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define K_ELEMS_PER_GRID 2048

#define PROJECT_DEF 1

struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float PhiMag;
};

/*The algorithm first computes the real and imaginary components of mu 
at each sample point in the trajectory space (k-space), 
then computes the real and imaginarycomponents of FHd at each voxel in the image space.*/


#if PROJECT_DEF
#define BLOCK_SIZE 512
#define K_VALS_GRID_SIZE (BLOCK_SIZE * 4)
__constant__ __device__ kValues const_kValues[K_VALS_GRID_SIZE];

//calculate mu at each sample point t
__global__ void ComputePhiMagKernel(int numK, float *phiR, float *phiI,
                                    float *phiMag)
{
	
// find the index of the voxel assigned to this thread
  unsigned int t = threadIdx.x + (blockIdx.x * blockDim.x);
  if (t < numK)
    phiMag[t] = (phiR[t] * phiR[t]) + (phiI[t] * phiI[t]);
}
/*The GPU-based implementation of the FHd algorithm uses constant memory caches to eliminate the
potential bottleneck posed by memory bandwidth and latency. 

The scan data is divided into many tiles, the host CPU loads the corresponding subset of sample points into constant memory 
before executing the cmpFhD function. 

Each thread computes a partial sum for a single element of FHd by iterating over all the sample points in the tile.

This optimization significantly increases the ratio of FP operations to global memory accesses.*/
//calculate FHd on one voxel//
__global__ void ComputeQKernel(int numK, int numX,
                               float *x_d, float *y_d, float *z_d,
                               float *Qr_d, float *Qi_d)
{ 
// find the index of the voxel assigned to this thread
  unsigned int t = threadIdx.x + (blockIdx.x * blockDim.x);

  if (t >= numX)
    return;
//register allocate voxel inputs and outputs
  float x_l = x_d[t];
  float y_l = y_d[t];
  float z_l = z_d[t];
  float Qracc = 0.0f;
  float Qiacc = 0.0f;
  float phi = 0.0f;

  float expArg;
  int idx = 0;
	

  if (numK % 2) {	  
    /* if numK is odd */
	// e^2pi*km*xn  
    expArg = PIx2 * (const_kValues[idx].Kx * x_l +
                     const_kValues[idx].Ky * y_l +
                     const_kValues[idx].Kz * z_l);
    phi = const_kValues[idx].PhiMag;
    /*First, instead of fath math, changing cos() and sin() function to hardware versions: __sin() and __cos(). 
     Because CUDA offers hardware implementations of mathematic functions that 
     provide much higher throughput than their software counterparts, but it will 
     reduced accuracy when switching from software functions to hardware functions.
     So must carefully.*/  
    /*failed to use five element taylor */
    Qracc += phi * _cos(expArg);
    Qiacc += phi * _sin(expArg);
    idx++;
  }

  for (; idx < numK; idx++) {
    /* using thread coarsening technique */
   //const_kValues(sample data)is held in costant memory
    expArg = PIx2 * (const_kValues[idx].Kx * x_l +
                     const_kValues[idx].Ky * y_l +
                     const_kValues[idx].Kz * z_l);

    phi = const_kValues[idx].PhiMag;
    Qracc += phi * _cos(expArg);
    Qiacc += phi * _sin(expArg);

    idx++;
    expArg = PIx2 * (const_kValues[idx].Kx * x_l +
                     const_kValues[idx].Ky * y_l +
                     const_kValues[idx].Kz * z_l);

    phi = const_kValues[idx].PhiMag;
   /*hardware versions: __sin() and __cos()*/
    Qracc += phi * _cos(expArg);
    Qiacc += phi * _sin(expArg);
  }
  Qr_d[t] += Qracc;
  Qi_d[t] += Qiacc;
}

void ComputePhiMagGPU(int numK, float* phiR_d, float* phiI_d,
                      float* phiMag_d)
{
  unsigned int numBlocks = ((numK - 1) / BLOCK_SIZE) + 1;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  ComputePhiMagKernel<<<dimGrid, dimBlock>>>(numK, phiR_d, phiI_d, phiMag_d);
}

void ComputeQGPU(int numK, int numX, struct kValues *kVals,
                 float *x_d, float *y_d, float *z_d, float *Qr_d, float *Qi_d)
{
  unsigned int size_to_cover = K_VALS_GRID_SIZE;
  unsigned int n_iter = ((numK - 1) / K_VALS_GRID_SIZE) + 1;
  struct kValues *ptr = kVals;

  unsigned int numBlocks = ((numX - 1) / BLOCK_SIZE) + 1;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  //printf("size : %d\n", sizeof(struct kValues));

  for (int iter = 0; iter < n_iter; iter++) {
    size_to_cover = MIN(K_VALS_GRID_SIZE, numK - (iter * K_VALS_GRID_SIZE));
    //printf("size to cover:%d, iter:%d, ptr:%u\n", size_to_cover, iter, ptr);
    if (size_to_cover) {
        hipMemcpyToSymbol(HIP_SYMBOL(const_kValues), ptr, size_to_cover * sizeof(struct kValues), 0);
        ComputeQKernel<<<dimGrid, dimBlock>>>(size_to_cover, numX, x_d, y_d, z_d, Qr_d, Qi_d);
        if (hipSuccess != hipDeviceSynchronize()) {
            printf("iter: %d ERROR!!!!!!\n", iter);
        }
    }
    ptr += size_to_cover;
  }

}

#else

inline
void
ComputePhiMagCPU(int numK,
                 float* phiR, float* phiI,
                 float* __restrict__ phiMag) {
  int indexK = 0;
  for (indexK = 0; indexK < numK; indexK++) {
    float real = phiR[indexK];
    float imag = phiI[indexK];
    phiMag[indexK] = real*real + imag*imag;
  }
}

inline
void
ComputeQCPU(int numK, int numX,
            struct kValues *kVals,
            float* x, float* y, float* z,
            float *__restrict__ Qr, float *__restrict__ Qi) {
  float expArg;
  float cosArg;
  float sinArg;

  int indexK, indexX;

  // Loop over the space and frequency domains.
  // Generally, numX > numK.
  // Since loops are not tiled, it's better that the loop with the smaller
  // cache footprint be innermost.
  for (indexX = 0; indexX < numX; indexX++) {

    // Sum the contributions to this point over all frequencies
    float Qracc = 0.0f;
    float Qiacc = 0.0f;
    for (indexK = 0; indexK < numK; indexK++) {
      expArg = PIx2 * (kVals[indexK].Kx * x[indexX] +
                       kVals[indexK].Ky * y[indexX] +
                       kVals[indexK].Kz * z[indexX]);

      cosArg = cosf(expArg);
      sinArg = sinf(expArg);

      float phi = kVals[indexK].PhiMag;
      Qracc += phi * cosArg;
      Qiacc += phi * sinArg;
    }
    Qr[indexX] = Qracc;
    Qi[indexX] = Qiacc;
  }
}

#endif

void createDataStructsCPU(int numK, int numX, float** phiMag,
	 float** Qr, float** Qi)
{
  *phiMag = (float* ) memalign(16, numK * sizeof(float));
  *Qr = (float*) memalign(16, numX * sizeof (float));
  memset((void *)*Qr, 0, numX * sizeof(float));
  *Qi = (float*) memalign(16, numX * sizeof (float));
  memset((void *)*Qi, 0, numX * sizeof(float));
}
